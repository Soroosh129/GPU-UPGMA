
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
//#include <common\book.h>

#define DIM 512
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;

#define min(a,b) (a<b)?a:b


__global__ void kernel(float *index,int *min_holder)
{
	__shared__ float tmp[DIM];
	int idx = threadIdx.x+blockIdx.x*blockDim.x;
	int local_index = threadIdx.x;
	int row_idx = blockIdx.x;
	__shared__ int min_index[DIM];




	int size = DIM/2;

	tmp[local_index] = index[idx];
	min_index[local_index] = local_index;
	__syncthreads();
	while(size)
	{
		if(local_index<size)
		{
			if(tmp[local_index+size]<tmp[local_index])
			{
				tmp[local_index]= tmp[local_index+size];
				min_index[local_index] = min_index[local_index+size];
			}
		}
		size/=2;
		__syncthreads();
	}

	if(local_index==0)
	{
		min_holder[row_idx] = min_index[0];
	}

}


int main()
{
	char file_name[255];// = "in.txt";
	ofstream fout("out.txt");
	cout<<"Please enter the file path to the distance matrix: ";
	cin.getline(file_name,255);
	std::vector<char> buffer(64 * 1024 * 1024);
	fstream fin;
	fin.rdbuf()->pubsetbuf(&buffer[0],buffer.size());
	fin.open(file_name);
	//cudaDeviceProp deviceProp;
	//cudaGetDeviceProperties(&deviceProp, 0);
	//cout<<deviceProp.name<<" has compute capability "<<deviceProp.major<<","<< deviceProp.minor<<endl;
	int size = INT_MIN;

	int r=0,c=0;
	fin>>size;

	int pitch=ceil((double)size/DIM);
	float *indexs=new float[size*size];
	int *min_holder = new int[size*pitch];
	float *indexes_d;
	int *min_holder_d;
	hipMalloc(&indexes_d,size*size*sizeof(float));
	hipMalloc(&min_holder_d,(size*pitch)*sizeof(int));
	bool *mark = new bool[size+1];

	for(int i=0; i<2000; i++)
	{
		indexs[i]=INT_MAX;

	}

	for(int i=0; i<size+1; i++)
		mark[i]=true;
	r=c=0;
	char tmp[255];
	cout<<"Reading input file";
	fin>>tmp;
	//cout<<tmp;
	while(1)
	{
		/*fin>>r>>c;
		r--;
		c--;*/

		fin>>indexs[r*size+c];
		c++; //:D
		//cout<<".";
		if(c==size)
		{
			mark[r]=false;
			r++;
			c=0;
			//cout<<endl;
			if(r<size)
			{
				fin>>tmp;
			}
			else
				break;
		}
	}

	cout<<" ..."<<endl;
	//cout<<size<<endl;
	//size--;
	int index=0;
	int handler=size;


	float min;
	float time;
	float time_total=0;
	cout<<"Working ";
	dim3 blocks(size*pitch);
	dim3 threads(512);
	while(handler)
	{
		cout<<".";
		min= INT_MAX;
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);

		//GPU code
		hipMemcpy(indexes_d,indexs,size*size*sizeof(float),hipMemcpyHostToDevice);
		kernel<<<blocks,threads>>>(indexes_d,min_holder_d);
		gpuErrchk(hipMemcpy(min_holder,min_holder_d,(size*pitch)*sizeof(int),hipMemcpyDeviceToHost));// end of GPU code
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time,start,stop);
		time_total+=time;
		if(time==0)
		{
			cout<<"\nSomething went wrong on GPU."<<endl;
			exit(0);
		}
		//cout<<"Time this round: "<<time<<endl;
		//for(int i=0; i<size*size ; i ++ )
		//cout<<i<<": "<<indexs[i]<<"  ";
		//cout<<endl;
		//getwchar();
		bool flag=false;
		int trow=-1;
		int row=0;
		int col=0;
		for(int k=0; k<size*pitch; k++)
		{
			if((k%(pitch))==0)
				trow++;
			int i = trow*size + min_holder[k];
			if(indexs[i]<min)
			{
				min=indexs[i];
				col = pitch*DIM+min_holder[k];
				row = trow;
				flag=true;
			}

		}

		//cout<<min<<endl;
		if(flag)
		{
			//cout<<row+1<<endl;
			fout<<row+1<<endl;
			//cout<<col+1<<endl;
			fout<<col+1<<endl;
		}
		//merging two rows and columns

		for(int i=0; i<size; i++)
		{
			indexs[col*size+i]= indexs[row*size+i]=(indexs[row*size+i]+indexs[col*size+i])/2;
			indexs[i*size+row]= indexs[i*size+col]=(indexs[i*size+row]+indexs[i*size+col])/2;
			indexs[i*size+i]=INT_MAX;

		}

		indexs[row*size+col] = indexs[col*size+row] = INT_MAX;
		handler--;
	}
	cout<<"\nTime: "<<time_total<<"ms"<<endl;
	cout<<"Press Enter to exit.";
	getchar();
	return 0;
}
