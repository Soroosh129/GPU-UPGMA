//#include <iostream>
//#include <fstream>
//#include <vector>
//#include <common/book.h>
//#include <limits.h>
//#include <fstream>
//#include <time.h>
//
//
//#define MAX 25000000
//
//using namespace std;
//
//__global__ void minimum(int *elements, int sz) {
//	__shared__ int tmp[4096];
//	int idx_x = threadIdx.x + blockIdx.x * blockDim.x;
//	int idx_y = threadIdx.y+blockIdx.y*blockDim.y;
//	int idx = idx_x+idx_y*blockDim.x*gridDim.x;
//	int local_index = threadIdx.x;
//	int row_idx = blockIdx.x * blockDim.x;
//
//	tmp[local_index] = elements[idx];
//
//	__syncthreads();
//
//	int size = (blockDim.x) / 2;
//	if (idx < sz)
//		while (size) {
//			if (local_index < size) {
//				if (tmp[local_index + size] <= tmp[local_index]) {
//					tmp[local_index] = tmp[local_index + size];
//				}
//			}
//			size /= 2;
//			__syncthreads();
//		}
//
//	if (local_index == 0) {
//		elements[row_idx] = tmp[0];
//	}
//
//}
//
//__global__ void minimum_with_index(int *elements, int *indexes, int sz) {
//	__shared__ int elements_shared[2048];
//	__shared__ int indexes_shared[2048];
//	int idx_x = threadIdx.x + blockIdx.x * blockDim.x;
//	int idx_y = threadIdx.y+blockIdx.y*blockDim.y;
//	int idx = idx_x+idx_y*blockDim.x*gridDim.x;
//	int local_index = threadIdx.x;
//	int row_idx = blockIdx.x * blockDim.x;
//
//	elements_shared[local_index] = elements[idx];
//	indexes_shared[local_index] = idx;
//
//	__syncthreads();
//
//	int size = (blockDim.x) / 2;
//	if (idx < sz)
//		while (size) {
//			if (local_index < size) {
//				if (elements_shared[local_index + size]
//						<= elements_shared[local_index]) {
//					elements_shared[local_index] = elements_shared[local_index
//							+ size];
//					indexes_shared[local_index] = indexes_shared[local_index
//							+ size];
//				}
//			}
//			size /= 2;
//			__syncthreads();
//		}
//
//	if (local_index == 0) {
//		elements[row_idx] = elements_shared[0];
//		indexes[row_idx] = indexes_shared[0];
//	}
//
//}
//
//__global__ void minimum_with_index_N(int *elements, int *indexes, int sz) {
//	__shared__ int elements_shared[2048];
//	__shared__ int indexes_shared[2048];
//	int idx_x = threadIdx.x + blockIdx.x * blockDim.x;
//	int idx_y = threadIdx.y+blockIdx.y*blockDim.y;
//	int idx = idx_x+idx_y*blockDim.x*gridDim.x;
//	int local_index = threadIdx.x;
//	int row_idx = blockIdx.x * blockDim.x;
//
//	elements_shared[local_index] = elements[idx];
//	indexes_shared[local_index] = idx;
//
//	__syncthreads();
//
//	int size = (blockDim.x) / 2;
//	if (idx < sz)
//		while (size) {
//			if (local_index < size) {
//				if (elements_shared[local_index + size]
//						<= elements_shared[local_index]) {
//					elements_shared[local_index] = elements_shared[local_index
//							+ size];
//					indexes_shared[local_index] = indexes_shared[local_index
//							+ size];
//				}
//			}
//			size /= 2;
//			__syncthreads();
//		}
//
//	if (local_index == 0) {
//		elements[row_idx] = elements_shared[0];
//		indexes[blockIdx.x] = indexes_shared[0];
//	}
//
//}
//
//int main() {
//	ofstream fout("out.txt");
//	cudaDeviceProp deviceProp;
//	cudaGetDeviceProperties(&deviceProp, 0);
//	cout<<deviceProp.name<<" has compute capability "<<deviceProp.major<<","<< deviceProp.minor<<endl<<"Shared Memory available: "<<deviceProp.sharedMemPerBlock<<endl;
//	int size = MAX;
//
//	int *elements_host = new int[size * size];
//	int *elements_device;
//	cudaMalloc(&elements_device, size * size * sizeof(int));
//
//	int *indexes_host = new int[size * size];
//	int *indexes_device;
//	cudaMalloc(&indexes_device, size * size * sizeof(int));
//
//	for (int i = 0; i < 2000; i++) {
//		elements_host[i] = INT_MAX;
//		indexes_host[i] = i; //for fun
//
//	}
//
//	float time;
//	int time_total = 0;
//
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//
//
//	cudaDeviceSynchronize();
//	dim3 blocks(size/512,size/512);
//	dim3 threads(512,512);
//	sleep(3);
//	size=0;
//	while (size <= MAX) {
//
//		fout<<size<<"\t";
//
//
//		cudaEventRecord(start, 0);
//		//GPU code
//		cudaMemcpy((void *) elements_device, elements_host,
//				size * size * sizeof(int), cudaMemcpyHostToDevice);
//		minimum<<<blocks, threads>>>(elements_device, size * size);
//		cudaMemcpy(elements_host, (void *) elements_device,
//				size * size * sizeof(int), cudaMemcpyDeviceToHost); // end of GPU code
//		cudaEventRecord(stop, 0);
//		cudaEventSynchronize(stop);
//		cudaEventElapsedTime(&time, start, stop);
//		cudaDeviceSynchronize();
//
//		fout << time<<"\t";
//		///////////////////////////////////////////////////////////////////////////
//		cudaEventRecord(start, 0);
//		//GPU code
//		cudaMemcpy((void *) elements_device, elements_host,
//				size * size * sizeof(int), cudaMemcpyHostToDevice);
//		minimum_with_index<<<blocks, threads>>>(elements_device, indexes_device,
//				size * size);
//		cudaMemcpy(elements_host, (void *) elements_device,
//				size * size * sizeof(int), cudaMemcpyDeviceToHost);
//		cudaMemcpy(indexes_host, (void *) indexes_device,
//				size * size * sizeof(int), cudaMemcpyDeviceToHost); // end of GPU code
//		cudaEventRecord(stop, 0);
//		cudaEventSynchronize(stop);
//		cudaEventElapsedTime(&time, start, stop);
//		cudaDeviceSynchronize();
//
//
//		fout << time<<"\t";
//		////////////////////////////////////////////////////////////////////////////
//		cudaEventRecord(start, 0);
//		//GPU code
//		cudaMemcpy((void *) elements_device, elements_host,
//				size * size * sizeof(int), cudaMemcpyHostToDevice);
//		minimum_with_index_N<<<blocks, threads>>>(elements_device, indexes_device,
//				size * size);
//		cudaMemcpy(elements_host, (void *) elements_device,
//				size * size * sizeof(int), cudaMemcpyDeviceToHost);
//		cudaMemcpy(indexes_host, (void *) indexes_device,
//				size * sizeof(int), cudaMemcpyDeviceToHost); // end of GPU code
//		cudaEventRecord(stop, 0);
//		cudaEventSynchronize(stop);
//		cudaEventElapsedTime(&time, start, stop);
//		time_total += time;
//		cudaDeviceSynchronize();
//
//
//		fout << time<<"\n";
//
//		size += 5000000;
//
//	}
//	return 0;
//}

#include <hip/hip_runtime.h>
#include <stdio.h>

const int N=10;

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}


int main(){

int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    for (int i=0; i<N; i++) {
        a[i] = -i; b[i] = i * i;
    }
    hipMemcpy ( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy ( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );

    add<<<1,N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost );

    for (int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i],b[i],c[i]);
    }

    hipFree (dev_a); hipFree (dev_b); hipFree (dev_c);

    return 0;

}
